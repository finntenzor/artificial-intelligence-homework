#include "hip/hip_runtime.h"
/**
 * layer_core_pooling.cu
 */

#include "layer_run.h"
#include "hip/hip_runtime.h"

// // ��һ��Ȩ��
// __device__ int layerGetPoolingWeightIndex(layer_schema_t* schema, int channelIndex, int rowIndex, int colIndex) {
//     // ����1��û��ƫ��
//     return (channelIndex * schema->inputHeight + rowIndex) * schema->inputWidth + colIndex;
// }

// __global__ void layerDevPredictPooling1(layer_schema_t schema, int batchSize) {
//     // �����ܸ��� = operationWidth * operationHeight * inputDepth
//     double* input = schema.predictInput;
//     double* args = schema.weights;

//     int inputRowBegin = (threadIdx.x / schema.operationHeight) * schema.operationHeight;
//     int inputColBegin = (threadIdx.y / schema.operationWidth) * schema.operationWidth;
//     int inputRowEnd = inputRowBegin + schema.operationHeight;
//     int inputColEnd = inputColBegin + schema.operationWidth;
//     int weightIndex = layerGetPoolingWeightIndex(&schema, blockIdx.y, threadIdx.x, threadIdx.y);

//     int maxCount = 0;
//     for (int b = 0; b < batchSize; b++) {
//         int maxRow = inputRowBegin;
//         int maxCol = inputColBegin;
//         for (int i = inputRowBegin; i < inputRowEnd; i++) { // �����к�ƫ����
//             for (int j = inputColBegin; j < inputColEnd; j++) { // �����к�ƫ����
//                 if (i < 0 || i >= schema.inputHeight || j < 0 || j >= schema.inputWidth) {
//                     continue;
//                 }
//                 int maxIndex = layerGetInputIndex(&schema, b, blockIdx.y, maxRow, maxCol);
//                 int inputIndex = layerGetInputIndex(&schema, b, blockIdx.y, i, j);
//                 if (input[inputIndex] > input[maxIndex]) {
//                     maxRow = i;
//                     maxCol = j;
//                 }
//             }
//         }
//         if (maxRow == threadIdx.x && maxCol == threadIdx.y) {
//             maxCount++;
//         }
//     }
//     args[weightIndex] = maxCount / batchSize;
// }

// int layerPredictPooling1(layer_schema_t* schema, int batchSize) {
//     dim3 gridSize(1, schema->inputDepth);
//     dim3 blockSize(schema->inputHeight, schema->inputWidth);
//     layerDevPredictPooling1<<<gridSize, blockSize>>>(*schema, batchSize);
//     return layerIfError(schema->layerIndex);
// }

// __global__ void layerDevPredictPooling2(layer_schema_t schema) {
//     // �����ܸ��� = operationWidth * operationHeight * inputDepth
//     double* output = schema.predictOutput;
//     double* input = schema.predictInput;
//     double* args = schema.weights;
//     int outputIndex = layerGetCurrentOutputIndex(&schema);

//     int inputRowBegin = threadIdx.x * schema.operationHeight;
//     int inputColBegin = threadIdx.y * schema.operationWidth;
//     int inputRowEnd = inputRowBegin + schema.operationHeight;
//     int inputColEnd = inputColBegin + schema.operationWidth;

//     double z = 0;
//     for (int i = inputRowBegin; i < inputRowEnd; i++) { // �����к�ƫ����
//         for (int j = inputColBegin; j < inputColEnd; j++) { // �����к�ƫ����
//             if (i < 0 || i >= schema.inputHeight || j < 0 || j >= schema.inputWidth) {
//                 continue;
//             }
//             int inputIndex = layerGetPoolingWeightIndex(&schema, blockIdx.y, i, j);
//             int weightIndex = layerGetPoolingWeightIndex(&schema, blockIdx.y, i, j);
//             z += input[inputIndex] * args[weightIndex];
//         }
//     }
//     output[outputIndex] = z;
// }

// int layerPredictPooling2(layer_schema_t* schema, int batchSize) {
//     dim3 gridSize(batchSize, schema->outputDepth);
//     dim3 blockSize(schema->outputHeight, schema->outputWidth);
//     layerDevPredictPooling2<<<gridSize, blockSize>>>(*schema);
//     return layerIfError(schema->layerIndex);
// }

// int layerPredictPooling(layer_schema_t* schema, int batchSize) {
//     int ret = layerPredictPooling1(schema, batchSize);
//     return ret || layerPredictPooling2(schema, batchSize);
// }

__global__ void layerDevPredictPooling1(double* output, double* input,
    int windowHeight, int windowWidth,
    int rowStep, int colStep,
    int rowBasis, int colBasis,
    int inputSize, int inputChannelSize, int inputHeight, int inputWidth,
    int outputSize, int outputChannelSize, int outputHeight, int outputWidth
)
{
    double* inputBase = input + blockIdx.x * inputSize + blockIdx.y * inputChannelSize;
    double* outputBase = output + blockIdx.x * outputSize + blockIdx.y * outputChannelSize;
    int inputRowBegin = threadIdx.x * rowStep + rowBasis;
    int inputColBegin = threadIdx.y * colStep + colBasis;
    int inputRowEnd = inputRowBegin + windowHeight;
    int inputColEnd = inputColBegin + windowWidth;

    double max = inputBase[0];
    for (int i = inputRowBegin; i < inputRowEnd; i++) {
        for (int j = inputColBegin; j < inputColEnd; j++) {
            if (i < 0 || i >= inputHeight || j < 0 || inputWidth) continue;
            double curr = inputBase[i * inputWidth + j];
            if (curr > max) {
                max = curr;
            }
        }
    }
    outputBase[threadIdx.x * outputWidth + threadIdx.y] = max;
}

int layerPredictPooling1(layer_schema_t* schema, int batchSize) {
    int inputSize = schema->inputDepth * schema->inputHeight * schema->inputWidth;
    int outputSize = schema->outputDepth * schema->outputHeight * schema->outputWidth;
    int inputChannelSize = schema->inputHeight * schema->inputWidth;
    int outputChannelSize = schema->outputHeight * schema->outputWidth;
    dim3 gridSize(batchSize, schema->outputDepth); // ����������Ӧ�����
    dim3 blockSize(schema->outputHeight, schema->outputWidth);
    layerDevPredictPooling1<<<gridSize, blockSize>>>(schema->predictOutput, schema->predictInput,
        schema->operationHeight, schema->operationWidth,
        schema->operationRowStep, schema->operationColStep,
        schema->operationRowBasis, schema->operationColBasis,
        inputSize, inputChannelSize, schema->inputHeight, schema->inputWidth,
        outputSize, outputChannelSize, schema->outputHeight, schema->outputWidth
    );
    return layerIfError(schema->layerIndex);
}

int layerPredictPooling(layer_schema_t* schema, int batchSize) {
    return layerPredictPooling1(schema, batchSize);
    // return ret || layerPredictPooling2(schema, batchSize);
}

__global__ void layerDevTrainPooling1(double* trainOutput, double* trainInput, double* input,
    int windowHeight, int windowWidth,
    int rowStep, int colStep,
    int rowBasis, int colBasis,
    int inputSize, int inputChannelSize, int inputHeight, int inputWidth,
    int outputSize, int outputChannelSize, int outputHeight, int outputWidth
)
{
    // double* inputBase = input + blockIdx.x * inputSize + blockIdx.y * inputChannelSize;
    // double* outputBase = output + blockIdx.x * outputSize + blockIdx.y * outputChannelSize;
    int inputRowBegin = threadIdx.x * rowStep + rowBasis - windowHeight + 1;
    int inputColBegin = threadIdx.y * colStep + colBasis - windowWidth + 1;
    int inputRowEnd = inputRowBegin + windowHeight;
    int inputColEnd = inputColBegin + windowWidth;

    // double max = inputBase[0];
    // for (int i = inputRowBegin; i < inputRowEnd; i++) {
    //     for (int j = inputColBegin; j < inputColEnd; j++) {
    //         if (i < 0 || i >= inputHeight || j < 0 || inputWidth) continue;
    //         double curr = inputBase[i * inputWidth + j];
    //         if (curr > max) {
    //             max = curr;
    //         }
    //     }
    // }
    // outputBase[threadIdx.x * outputWidth + threadIdx.y] = max;
}

int layeyTrainPooling1(layer_schema_t* schema, int batchSize) {
    int inputSize = schema->inputDepth * schema->inputHeight * schema->inputWidth;
    int outputSize = schema->outputDepth * schema->outputHeight * schema->outputWidth;
    int inputChannelSize = schema->inputHeight * schema->inputWidth;
    int outputChannelSize = schema->outputHeight * schema->outputWidth;
    dim3 gridSize(batchSize, schema->inputDepth);
    dim3 blockSize(schema->inputHeight, schema->inputWidth);
    layerDevTrainPooling1<<<gridSize, blockSize>>>(schema->trainOutput, schema->trainInput, schema->predictInput,
        schema->operationHeight, schema->operationWidth,
        schema->operationRowStep, schema->operationColStep,
        schema->operationRowBasis, schema->operationColBasis,
        inputSize, inputChannelSize, schema->inputHeight, schema->inputWidth,
        outputSize, outputChannelSize, schema->outputHeight, schema->outputWidth
    );
    return layerIfError(schema->layerIndex);
}

int layerTrainPooling(layer_schema_t* schema, int batchSize) {
    return layeyTrainPooling1(schema, batchSize);
}
