#include "hip/hip_runtime.h"
/**
 * layer_core_scale.cu
 */

#include "layer_run.h"
#include "hip/hip_runtime.h"

__global__ void layerDevPredictScale1(double* input, double* output, int outputSize) {
    int inputBegin = blockIdx.x * outputSize;

    double value = 0;
    // �ҳ����ֵ
    for (int i = 0; i < outputSize; i++) {
        if (input[inputBegin + i] > value) {
            value = input[inputBegin + i];
        }
    }
    // �������ֵ
    output[blockIdx.x] = value;
}

int layerPredictScale1(layer_schema_t* schema, int batchSize) {
    int outputSize = schema->outputDepth * schema->outputHeight * schema->outputWidth;
    dim3 gridSize(batchSize);
    dim3 blockSize(1);
    layerDevPredictScale1<<<gridSize, blockSize>>>(schema->predictInput, schema->predictTemp, outputSize);
    return layerIfError(schema->layerIndex);
}

__global__ void layerDevPredictScale2(double* input, double* maxz, double* output, int outputSize) {
    int index = blockIdx.x * outputSize + threadIdx.x;

    // �����������Լ����ڿ�ĵ���Сֵ
    if (maxz[blockIdx.x] == 0) {
        printf("layerDevPredictScale2 ZERO: blockIdx.x = %d, threadIdx.x = %d, index = %d\n", blockIdx.x, threadIdx.x, index);
    }
    output[index] = input[index] / maxz[blockIdx.x];
}

int layerPredictScale2(layer_schema_t* schema, int batchSize) {
    int outputSize = schema->outputDepth * schema->outputHeight * schema->outputWidth;
    dim3 gridSize(batchSize);
    dim3 blockSize(outputSize);
    layerDevPredictScale2<<<gridSize, blockSize>>>(schema->predictInput, schema->predictTemp, schema->predictOutput, outputSize);
    return layerIfError(schema->layerIndex);
}

int layerPredictScale(layer_schema_t* schema, int batchSize) {
    int ret = 0;
    ret = ret || layerPredictScale1(schema, batchSize);
    ret = ret || layerPredictScale2(schema, batchSize);
    return ret;
}

__global__ void layerDevTrainScale(double* input, double* maxz, double* output, int outputSize, int layerIndex) {
    int index = blockIdx.x * outputSize + threadIdx.x;

    // �������ں�һ�㵼������ϵ��
    // input��trainInput
    if (maxz[blockIdx.x] == 0) {
        printf("layerDevTrainScale ZERO: blockIdx.x = %d, threadIdx.x = %d, index = %d\n", blockIdx.x, threadIdx.x, index);
    }
    output[index] = input[index] / maxz[blockIdx.x];

    // DEBUG
    // if (layerIndex == 6) {
    //     printf("blockIdx.x = %d, threadIdx.x = %d, index = %d, input at %p, input = %.12lf, maxz = %.12lf, output = %.12lf\n", blockIdx.x, threadIdx.x, index, input, input[index], maxz[blockIdx.x], output[index]);
    // }
}

int layerTrainScale(layer_schema_t* schema, int batchSize) {
    int outputSize = schema->outputDepth * schema->outputHeight * schema->outputWidth;
    dim3 gridSize(batchSize);
    dim3 blockSize(outputSize);
    layerDevTrainScale<<<gridSize, blockSize>>>(schema->trainInput, schema->predictTemp, schema->trainOutput, outputSize, schema->layerIndex);
    return layerIfError(schema->layerIndex);
}
