#include "hip/hip_runtime.h"
/**
 * layer_core_input.cu
 */

#include "layer_run.h"
#include "hip/hip_runtime.h"

__global__ void layerDevPredictInput(layer_schema_t schema, unsigned char* input) {
    // �����Ӧ���������������ȣ����ﲻ�����
    // ����ͨ����Ӧ������1�����ﲻ�����
    double* output = schema.predictOutput;
    int inputIndex = layerGetInputIndex(&schema, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
    int outputIndex = layerGetCurrentOutputIndex(&schema);
    output[outputIndex] = (input[inputIndex] + 1) / 256.0;

    // DEBUG
    // printf("&output = %p, blockIdx.x = %d, blockIdx.y = %d, threadIdx.x = %d, threadIdx.y = %d, inputIndex = %d, ouputIndex = %d, input = %d, output = %lf\n", output, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, inputIndex, outputIndex, input[inputIndex], output[outputIndex]);
}

int layerPredictInput(layer_schema_t* schema, int batchSize, unsigned char* input) {
    dim3 gridSize(batchSize, schema->outputDepth);
    dim3 blockSize(schema->outputHeight, schema->outputWidth);
    layerDevPredictInput<<<gridSize, blockSize>>>(*schema, input);
    return layerIfError(schema->layerIndex);
}
