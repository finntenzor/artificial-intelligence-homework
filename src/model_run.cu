#include "hip/hip_runtime.h"
/**
 * model_run.cu
 */

#include "model_run.h"
#include "hip/hip_runtime.h"

static int minInt(int a, int b) {
    return a < b ? a : b;
}

int modelIfErrorWithStatus(hipError_t cudaStatus, const char* str) {
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s, CUDA��Ϣ: %s\n", str, hipGetErrorString(cudaStatus));
        return 1;
    }
    return 0;
}

int modelIfError(const char* str) {
    hipError_t cudaStatus = hipSuccess;
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s, CUDA��Ϣ: %s\n", str, hipGetErrorString(cudaStatus));
        return 1;
    }
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)  {
        fprintf(stderr, "%s, CUDA��Ϣ: %s\n", str, hipGetErrorString(cudaStatus));
        return 1;
    }
    return 0;
}

int modelCalcGridThreadCount(int total, int* block, int* thread) {
    hipError_t cudaStatus;
    hipDeviceProp_t devProp;
    int currentDevice;
    int maxThread = -1;
    int q, r;
    if (total <= 0) {
        fprintf(stderr, "�ܸ���������������ʵ������ %d\n", total);
        return 1;
    }
    cudaStatus = hipGetDevice(&currentDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "�޷���ȡ��ǰ���е�GPU�豸��\n");
        return 1;
    }
    cudaStatus = hipGetDeviceProperties(&devProp, currentDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "�޷���ȡ��%d���豸���豸��Ϣ\n", currentDevice);
        return 1;
    }
    maxThread = devProp.maxThreadsPerBlock;
    q = total / maxThread;
    r = total % maxThread;
    if (r > 0) q++;
    *block = q;
    *thread = maxThread;
    return 0;
}

int modelGetBatchCount(model_schema_t* mem) {
    int batchCount = mem->inputCount / mem->batchSize;
    if (mem->batchSize * batchCount < mem->inputCount) {
        batchCount++;
    }
    return batchCount;
}

int modelRunBatch(model_schema_t* mem, int offset) {
    int ret = 0;
    int inputBlockSize = modelGetInputBlockSize(mem);
    int outputBlockSize = modelGetOutputBlockSize(mem);
    int labelsBlockSize = modelGetLabelsBlockSize(mem);
    int batchSize = minInt(mem->inputCount - offset, mem->batchSize);
    unsigned char* input = mem->input + offset * inputBlockSize;
    unsigned char* output = mem->output + offset * outputBlockSize;
    unsigned char* labels = mem->labels + offset * labelsBlockSize;

    if (batchSize == 0) {
        return 0;
    }

    // printf("RUN BATCH, offset = %d\n", offset);

    for (int i = 0; i < mem->schemaCount; i++) {
        // printf("RUN LAYER %d\n", i);
        layer_schema_t* schema = &mem->schemas[i];
        switch (schema->type) {
        case LAYER_TYPE_INPUT:
            ret = layerPredictInput(schema, batchSize, input);
            break;
        case LAYER_TYPE_CONVOLUTION:
            ret = layerPredictConvolution(schema, batchSize);
            break;
        case LAYER_TYPE_POOLING:
            ret = layerPredictPooling(schema, batchSize);
            break;
        case LAYER_TYPE_DENSE:
            ret = layerPredictDense(schema, batchSize);
            break;
        case LAYER_TYPE_SCALE:
            ret = layerPredictScale(schema, batchSize);
            break;
        case LAYER_TYPE_OUTPUT:
            ret = layerPredictOutput(schema, batchSize, output);
            ret = ret || layerCheckOutput(schema, batchSize, output, labels, mem->accuracyRate, mem->loss);
            break;
        }
        if (ret) break;
    }
    return ret;
}

int modelFetchAccuracyRate(model_schema_t* mem, double* acc) {
    hipError_t cudaStatus = hipSuccess;
    cudaStatus = hipMemcpy(acc, mem->accuracyRate, 1 * sizeof(double), hipMemcpyDeviceToHost);
    if (modelIfErrorWithStatus(cudaStatus, "�޷���׼ȷ�ʴ��Դ濽�����ڴ�")) return 1;
    return 0;
}

int modelFetchLoss(model_schema_t* mem, double* loss) {
    hipError_t cudaStatus = hipSuccess;
    cudaStatus = hipMemcpy(loss, mem->loss, 1 * sizeof(double), hipMemcpyDeviceToHost);
    if (modelIfErrorWithStatus(cudaStatus, "�޷�����ʧ���Դ濽�����ڴ�")) return 1;
    return 0;
}

// __global__ void modelDevGetBatchAccuracyCount(double* accuracyCount, unsigned char* output, unsigned char* labels, int size) {
//     int accCount = 0;
//     for (int i = 0; i < size; i++) {
//         int label = labels[i];
//         int predict = output[i];
//         if (predict == label) {
//             accCount++;
//         }
//     }
//     *accuracyCount = accCount;
// }

// int modelGetBatchAccuracy(model_schema_t* mem, int offset, double* accuracyCount, int* size) {
//     hipError_t cudaStatus = hipSuccess;
//     double acc = 0;
//     int outputBlockSize = modelGetOutputBlockSize(mem);
//     int labelsBlockSize = modelGetLabelsBlockSize(mem);
//     int batchSize = minInt(mem->inputCount - offset, mem->batchSize);
//     unsigned char* output = mem->output + offset * outputBlockSize;
//     unsigned char* labels = mem->labels + offset * labelsBlockSize;

//     if (batchSize == 0) {
//         return -1;
//     }

//     modelDevGetBatchAccuracyCount<<<1, 1>>>(mem->accuracyRate, output, labels, batchSize);
//     if (modelIfError("����ģ��׼ȷ��ʱ��������")) return 1;

//     cudaStatus = hipMemcpy(&acc, mem->accuracyRate, 1 * sizeof(double), hipMemcpyDeviceToHost);
//     if (modelIfErrorWithStatus(cudaStatus, "�޷���׼ȷ�ʴ��Դ濽�����ڴ�")) return 2;

//     *accuracyCount = acc;
//     *size = batchSize;
//     return 0;
// }

// double modelGetBatchAccuracyRate(model_schema_t* mem, int offset) {
//     double acc;
//     int size;
//     int ret = modelGetBatchAccuracy(mem, offset, &acc, &size);
//     return (ret) ? (-1) : (acc / size);
// }

// double modelGetAccuracyRate(model_schema_t* mem) {
//     int batchCount = modelGetBatchCount(mem);
//     double accCount = 0;
//     for (int i = 0; i < batchCount; i++) {
//         double acc;
//         int size;
//         int ret = modelGetBatchAccuracy(mem, mem->batchSize, &acc, &size);
//         if (ret) {
//             accCount = -1;
//             break;
//         } else {
//             accCount += acc;
//         }
//     }
//     return (accCount >= 0) ? (accCount / mem->inputCount) : (-1);
// }

int modelPredict(model_schema_t* mem) {
    int ret = 0;
    int batchCount = modelGetBatchCount(mem);
    for (int i = 0; i < batchCount; i++) {
        ret = ret || modelRunBatch(mem, i * mem->batchSize);
        if (ret) break;
    }
    return ret;
}

__global__ void modelDevClearDweights(double* output, int dwsize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < dwsize) {
        output[index] = 0;
    }
}

int modelClearDweights(model_schema_t* mem) {
    int dwsize = modelGetDweightsSize(mem);
    int block, thread;
    modelCalcGridThreadCount(dwsize, &block, &thread);
    dim3 gridSize(block);
    dim3 blockSize(thread);
    modelDevClearDweights<<<gridSize, blockSize>>>(mem->dweights, dwsize);
    if (modelIfError("���ģ��Ȩ�ر仯��ʱ��������")) return 1;
    return 0;
}

int modelTrainBatch(model_schema_t* mem, int offset) {
    int ret = 0;
    int labelsBlockSize = modelGetLabelsBlockSize(mem);
    int batchSize = minInt(mem->inputCount - offset, mem->batchSize);
    unsigned char* labels = mem->labels + offset * labelsBlockSize;

    if (batchSize == 0) {
        return 0;
    }

    for (int i = mem->schemaCount - 1; i >= 0 ; i--) {
        layer_schema_t* schema = &mem->schemas[i];
        switch (schema->type) {
        // case LAYER_TYPE_INPUT:
        //     ret = layerPredictInput(schema, batchSize, input);
        //     break;
        // case LAYER_TYPE_CONVOLUTION:
        //     ret = layerPredictConvolution(schema, batchSize);
        //     break;
        // case LAYER_TYPE_POOLING:
        //     ret = layerPredictPooling(schema, batchSize);
        //     break;
        case LAYER_TYPE_DENSE:
            ret = layerTrainDense(schema, batchSize);
            break;
        case LAYER_TYPE_SCALE:
            ret = layerTrainScale(schema, batchSize);
            break;
        case LAYER_TYPE_OUTPUT:
            ret = layerTrainOutput(schema, batchSize, labels);
            break;
        }
        if (ret) break;
    }
    return ret;
}

__global__ void modelDevApplyDweights(double studyRate, double* weights, double* dweights, int wsize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < wsize) {
        weights[index] -= studyRate * dweights[index];
        // printf("blockIdx.x = %d, threadIdx.x = %d, index = %d, studyRate = %lf, dweights = %lf, weights = %lf\n", blockIdx.x, threadIdx.x, index, studyRate, dweights[threadIdx.x], weights[threadIdx.x]);
    }
}

int modelApplyDweights(model_schema_t* mem, int offset) {
    int wsize = modelGetWeightsSize(mem);
    int block, thread;
    int batchSize = minInt(mem->inputCount - offset, mem->batchSize);
    modelCalcGridThreadCount(wsize, &block, &thread);
    dim3 gridSize(block);
    dim3 blockSize(thread);
    modelDevApplyDweights<<<gridSize, blockSize>>>(mem->studyRate, mem->weights, mem->dweights, wsize);
    if (modelIfError("����ģ��Ȩ��ʱ��������")) return 1;
    return 0;
}

int modelTrain(model_schema_t* mem, int (*batchCallback)(model_schema_t* mem, int batchIndex, int step)) {
    int ret = 0;
    int batchCount = modelGetBatchCount(mem);
    ret = modelClearDweights(mem);
    double accuracyRate = 0;
    double loss = 0;

    // TODO ��ep��Ϊloss�ж�
    for (int ep = 0; !ret && ep < 10; ep++) {
        for (int i = 0; !ret && i < batchCount; i++) {
            int offset = i * mem->batchSize;
            if (batchCallback != NULL) {
                ret = ret || (*batchCallback)(mem, i, 0);
            }
            ret = ret || modelRunBatch(mem, offset);
            if (batchCallback != NULL) {
                ret = ret || (*batchCallback)(mem, i, 1);
            }
            ret = ret || modelTrainBatch(mem, offset);
            if (batchCallback != NULL) {
                ret = ret || (*batchCallback)(mem, i, 2);
            }
            ret = ret || modelApplyDweights(mem, offset);
            if (batchCallback != NULL) {
                ret = ret || (*batchCallback)(mem, i, 3);
            }
            ret = ret || modelFetchAccuracyRate(mem, &accuracyRate);
            ret = ret || modelFetchLoss(mem, &loss);
            printf("��ǰ��ȷ�� = %10.6lf%%, ��ʧ = %.6lf\n", accuracyRate * 100, loss);
        }
    }
    return ret;
}
