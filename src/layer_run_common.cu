/**
 * layer_run.cu
 */

#include "layer_run.h"
#include "hip/hip_runtime.h"

/*
 * ����������Ӧ�±�
 * ���еĲ�ͳһ����ά���飬�ĸ�ά��������
 * ��� ÿ��batch����batchSize����
 * ͨ���� ���Ϊ��������-1
 * �к�
 * �к�
 */
__device__ int layerGetInputIndex(layer_schema_t* schema, int blockIndex, int channelIndex, int rowIndex, int colIndex) {
    return ((blockIndex * schema->inputDepth + channelIndex) * schema->inputHeight + rowIndex) * schema->inputWidth + colIndex;
}

__device__ int layerGetOutputIndex(layer_schema_t* schema, int blockIndex, int channelIndex, int rowIndex, int colIndex) {
    return ((blockIndex * schema->outputDepth + channelIndex) * schema->outputHeight + rowIndex) * schema->outputWidth + colIndex;
}

__device__ int layerGetCurrentOutputIndex(layer_schema_t* schema) {
    return layerGetOutputIndex(schema, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

int layerIfError(int layerIndex) {
    hipError_t cudaStatus = hipSuccess;
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "�� %d �㷢������CUDA��Ϣ: %s\n", layerIndex, hipGetErrorString(cudaStatus));
        return 1;
    }
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)  {
        fprintf(stderr, "�� %d �㷢������CUDA��Ϣ: %s\n", layerIndex, hipGetErrorString(cudaStatus));
        return 1;
    }
    return 0;
}
